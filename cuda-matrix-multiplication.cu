
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 2

typedef struct {
    int width;
    int height;
    int stride; 
    float* elements;
} Matrix;

__device__ Matrix getSubMatrix(Matrix X, int row, int col);
__device__ float getElement(const Matrix X, int row, int col);
__device__ void setElement(Matrix X, int row, int col, float value);

 __global__ void multipleMatrixesKernel(Matrix X, Matrix Y, Matrix Z) {

    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    Matrix sZ = getSubMatrix(Z, blockRow, blockCol);

    float Zvalue = 0;

    int row = threadIdx.y;
    int col = threadIdx.x;

    for (int m = 0; m < (X.width / BLOCK_SIZE); ++m) {

        Matrix sX = getSubMatrix(X, blockRow, m);

        Matrix sY = getSubMatrix(Y, m, blockCol);

        __shared__ float Xs[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Ys[BLOCK_SIZE][BLOCK_SIZE];

        Xs[row][col] = getElement(sX, row, col);
        Ys[row][col] = getElement(sY, row, col);

        __syncthreads();

        for (int e = 0; e < BLOCK_SIZE; ++e)
            Zvalue += Xs[row][e] * Ys[e][col];

        __syncthreads();
    }
    setElement(sZ, row, col, Zvalue);
}

void multipleMatrixes(const Matrix X, const Matrix Y, Matrix Z) { 
    Matrix d_X;
    d_X.width = d_X.stride = X.width; 
    d_X.height = X.height;
    size_t size = X.width * X.height * sizeof(float);
    hipMalloc(&d_X.elements, size);
    hipMemcpy(d_X.elements, X.elements, size, hipMemcpyHostToDevice);

    Matrix d_Y;
    d_Y.width = d_Y.stride = Y.width; 
    d_Y.height = Y.height;
    size = Y.width * Y.height * sizeof(float);
    hipMalloc(&d_Y.elements, size);
    hipMemcpy(d_Y.elements, Y.elements, size,
    hipMemcpyHostToDevice);

    Matrix d_Z;
    d_Z.width = d_Z.stride = Z.width; 
    d_Z.height = Z.height;
    size = Z.width * Z.height * sizeof(float);
    hipMalloc(&d_Z.elements, size);

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(Y.width / dimBlock.x, X.height / dimBlock.y);
    multipleMatrixesKernel<<<dimGrid, dimBlock>>>(d_X, d_Y, d_Z);

    hipMemcpy(Z.elements, d_Z.elements, size, hipMemcpyDeviceToHost);

    hipFree(d_X.elements);
    hipFree(d_Y.elements);
    hipFree(d_Z.elements);
}

__device__ Matrix getSubMatrix(Matrix X, int row, int col) {
    Matrix sX;
    sX.width = BLOCK_SIZE;
    sX.height = BLOCK_SIZE;
    sX.stride = X.stride;
    sX.elements = &X.elements[X.stride * BLOCK_SIZE * row + BLOCK_SIZE * col];
    return sX;
}

__device__ float getElement(const Matrix X, int row, int col) {
    return X.elements[row * X.stride + col];
}

__device__ void setElement(Matrix X, int row, int col, float value) {
    X.elements[row * X.stride + col] = value;
}

void fillMatrixRandomValues(float* matrix, int rows, int columns){
    for(int i = 0; i < rows; i++){
        for(int j = 0; j < columns; j++){
            matrix[i * columns + j] = rand() % 9 + 1;
        }
    }
}

void printMatrix(float* matrix, int rows, int columns){
    for(int i = 0; i < rows; i++){
        for(int j = 0; j < columns; j++){
            printf("%.2f ", matrix[i * columns + j]);
        }
        printf("\n");
    }
}

int main()
{
    int width = 4;
    int height = 4;
    int stride = 1;
    float *elementsX = (float *)malloc(width * height * sizeof(float));
    float *elementsY = (float *)malloc(width * height * sizeof(float));
    float *elementsZ = (float *)malloc(width * height * sizeof(float));

    fillMatrixRandomValues(elementsX, width, height);
    fillMatrixRandomValues(elementsY, width, height);

    Matrix X;
    X.width = width;
    X.height = height;
    X.stride = stride;
    X.elements = elementsX;

    Matrix Y;
    Y.width = width;
    Y.height = height;
    Y.stride = stride;
    Y.elements = elementsY;

    Matrix Z;
    Z.width = width;
    Z.height = height;
    Z.stride = stride;
    Z.elements = elementsZ;

    printMatrix(X.elements, width, height);
    printf("\n");
    printMatrix(Y.elements, width, height);
    printf("\n");

    multipleMatrixes(X, Y, Z);

    printMatrix(Z.elements, width, height);

    free(elementsX);
    free(elementsY);
    free(elementsZ);

    return 0;

}
